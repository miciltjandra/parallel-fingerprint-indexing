#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <chrono>
#include <thrust/sort.h>
#include "fingerprint_structure.hpp"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

const int BLOCKSIZE = 36;

// Constant weights
const float w1 = 0.16f;
const float w2 = 0.37f;
const float w3 = 0.16f;
const float w4 = 0.31f;

__host__ __device__ unsigned char dperiod_to_byte(float period) {
    float fresult = period/period_unit;
    unsigned char result = (char)fresult;
    return result;
}

__host__ __device__ float dbyte_to_period(unsigned char c) {
    float result = period_unit*(int)c;
    return result;
}

__host__ __device__ unsigned char dfrequency_to_byte(float frequency) {
    if (frequency == 0) {
        return dperiod_to_byte(frequency);
    } else {
        return dperiod_to_byte(1.0f/frequency);
    }
}

__host__ __device__ float dbyte_to_frequency(unsigned char c) {
    float result = dbyte_to_period(c);
    if (result == 0) return result;
    else return 1/result;
}

__device__ float dbyte_to_coherence(unsigned char c) {
    float result = (float)c/coherence_unit;
    return result;
}

__device__ float dbyte_to_orientation(unsigned char c) {
    float result = orientation_unit*(int)c;
    return result;
}

__global__ void calculate_s1_preparation(fingerprint* db, fingerprint* fp, float* s_sum, float* cos_sum, float* sin_sum) {
    __shared__ float ss, scos, ssin;
    int j = blockIdx.x;
    int i = threadIdx.x;
    if (i == 0) {
        ss = 0;
        scos = 0;
        ssin = 0;
    }
    __syncthreads();
    float s = dbyte_to_coherence(fp->local_coherence[i])*dbyte_to_coherence((db+j)->local_coherence[i]);
    float d = M_PI/180.0f * 2 * (dbyte_to_orientation(fp->local_orientation[i])-dbyte_to_orientation((db+j)->local_orientation[i]));
    float tcos = s*cos(d);
    float tsin = s*sin(d);

    atomicAdd(&ss, s);
    atomicAdd(&scos, tcos);
    atomicAdd(&ssin, tsin);
    __syncthreads();
    if (i == 0) {
        s_sum[j] = ss;
        cos_sum[j] = scos;
        sin_sum[j] = ssin;
    }
}

__global__ void calculate_s1(float* s_sum, float* cos_sum, float* sin_sum, float* result, int count) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    if (j<count) result[j] = sqrt(pow(cos_sum[j],2)+pow(sin_sum[j],2))/s_sum[j];
}

__global__ void get_best_core_s1(fingerprint* db, float* result, int* mapping) {
    int i = blockIdx.x;
    if ((db+i)->id%5 == 1) {
        int max_idx = i;
        for (int j=1 ; j<5 ; j++) {
            if ((db+i+j)->id%5 == 1) break;
            else {
                if (result[i+j] > result[max_idx]) {
                    max_idx = i+j;
                }
            }
        }
        mapping[((db+i)->id-1)/5] = max_idx;
    }
}

__global__ void calculate_s2(fingerprint* db, fingerprint* fp, float* result, int* mapping) {
    __shared__ float s_addition, s_absdiff;
    int j = mapping[blockIdx.x];
    int i = threadIdx.x;
    if (i == 0) {
        s_addition = 0.0f;
        s_absdiff = 0.0f;
    }
    float t_addition = dbyte_to_frequency(fp->local_frequency[i]) + dbyte_to_frequency((db+j)->local_frequency[i]);
    float t_absdiff = abs(dbyte_to_frequency(fp->local_frequency[i]) - dbyte_to_frequency((db+j)->local_frequency[i]));
    atomicAdd(&s_addition, t_addition);
    atomicAdd(&s_absdiff, t_absdiff);
    __syncthreads();
    if (i == 0) {
        result[blockIdx.x] = 1 - (s_absdiff/s_addition);
    }
}

__global__ void calculate_s3(fingerprint* db, fingerprint* fp, float* result, int* mapping) {
    int j = mapping[blockIdx.x];
    result[blockIdx.x] = 1 - (abs(dbyte_to_frequency(fp->avg_frequency)-dbyte_to_frequency((db+j)->avg_frequency))/max(dbyte_to_frequency(fp->avg_frequency), dbyte_to_frequency((db+j)->avg_frequency)));
}

__global__ void calculate_s4(fingerprint* db, fingerprint* fp, float* result, int* mapping) {
    int j = mapping[blockIdx.x];
    result[blockIdx.x] = 1-(abs(dbyte_to_orientation(fp->avg_orientation)-dbyte_to_orientation((db+j)->avg_orientation))/180.0f);
}

__global__ void calculate_s(float* s1, float* s2, float*s3, float* s4, float* result, int* mapping) {
    int i = blockIdx.x;
    result[i] = w1*s1[mapping[i]] + w2*s2[i] + w3*s3[i] + w4*s4[i];
}

__global__ void get_top_fingerprints(float* s, float* result, int* mapping) {
    int i = threadIdx.x;
    result[i] = s[mapping[i]];
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage : ./parallel_indexing fingerprint-to-be-searched fingerprint-db\n";
        return 0;
    }

    std::string fp_filename = argv[1];
    std::string db_filename = argv[2];

    // Read the fingerprint to be searched
    std::vector<struct fingerprint> fp;
    int count_fp = read_from_file(fp, fp_filename);

    // Read the database
    std::vector<struct fingerprint> db;
    int count_db = read_from_file(db, db_filename);
    std::cerr << "Fingerprint core database count : " << count_db << std::endl;

    std::cerr << "Last fingerprint ID : " << db[count_db-1].id << std::endl;
    int count_db_fingerprint = (db[count_db-1].id-1)/5+1;
    std::cerr << "Fingerprint database count : " << count_db_fingerprint << std::endl;

    auto timer_start = std::chrono::steady_clock::now();

    // Preparing memory
    fingerprint *d_fp, *d_db;
    std::vector<float> result(count_db_fingerprint, 0);
    float *d_s1_result, *d_s2_result, *d_s3_result, *d_s4_result, *d_result;
    
    hipMalloc((void **)&d_fp, sizeof(fingerprint));
    hipMalloc((void **)&d_db, count_db*sizeof(fingerprint));

    hipMalloc((void **)&d_s1_result, count_db*sizeof(float));
    hipMalloc((void **)&d_s2_result, count_db_fingerprint*sizeof(float));
    hipMalloc((void **)&d_s3_result, count_db_fingerprint*sizeof(float));
    hipMalloc((void **)&d_s4_result, count_db_fingerprint*sizeof(float));
    hipMalloc((void **)&d_result, count_db_fingerprint*sizeof(float));

    //Mapping for fingerprint to fingerprint core idx
    int *d_mapping;
    hipMalloc((void **)&d_mapping, count_db_fingerprint*sizeof(int));

    hipMemcpy(d_db, &db[0], count_db*sizeof(fingerprint), hipMemcpyHostToDevice);
    hipMemcpy(d_fp, &fp[0], sizeof(fingerprint), hipMemcpyHostToDevice);

    //Additional Memory for S1
    float *d_s_sum, *d_cos_sum, *d_sin_sum;
    hipMalloc((void **)&d_s_sum, count_db*sizeof(float));
    hipMalloc((void **)&d_cos_sum, count_db*sizeof(float));
    hipMalloc((void **)&d_sin_sum, count_db*sizeof(float));

    // S1
    calculate_s1_preparation<<<count_db,BLOCKSIZE>>>(d_db, d_fp, d_s_sum, d_cos_sum, d_sin_sum);
    calculate_s1<<<(count_db/256)+1, 256>>>(d_s_sum, d_cos_sum, d_sin_sum, d_s1_result, count_db);
    get_best_core_s1<<<count_db, 1>>>(d_db, d_s1_result, d_mapping);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    std::vector<int> mapping(count_db_fingerprint, 0);
    hipMemcpy(&mapping[0], d_mapping, count_db_fingerprint*sizeof(int), hipMemcpyDeviceToHost);
    // std::vector<float> s1_result;
    // s1_result.resize(count_db, 0);
    // hipMemcpy(&s1_result[0], d_s1_result, count_db*sizeof(float), hipMemcpyDeviceToHost);

    // S2
    // Only calculate for 1 core per fingerprint using mapping
    calculate_s2<<<count_db_fingerprint,BLOCKSIZE>>>(d_db, d_fp, d_s2_result, d_mapping);
    // hipMemcpy(&s2_result[0], d_s2_result, count_db_fingerprint*sizeof(float), hipMemcpyDeviceToHost);

    // S3
    calculate_s3<<<count_db_fingerprint,1>>>(d_db, d_fp, d_s3_result,d_mapping);
    // hipMemcpy(&s3_result[0], d_s3_result, count_db*sizeof(float), hipMemcpyDeviceToHost);

    // S4
    calculate_s4<<<count_db_fingerprint,1>>>(d_db, d_fp, d_s4_result, d_mapping);
    // hipMemcpy(&s4_result[0], d_s4_result, count_db*sizeof(float), hipMemcpyDeviceToHost);

    // S
    calculate_s<<<count_db_fingerprint, 1>>>(d_s1_result, d_s2_result, d_s3_result, d_s4_result, d_result, d_mapping);
    // hipMemcpy(&result[0], d_result, count_db_fingerprint*sizeof(float), hipMemcpyDeviceToHost);

    // ID for identifying fingerprint during sort
    int* ids = new int[count_db_fingerprint];
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        ids[i] = db[mapping[i]].id;
    }
    int* d_ids;
    hipMalloc((void **)&d_ids, count_db_fingerprint*sizeof(int));
    hipMemcpy(d_ids, &ids[0], count_db_fingerprint*sizeof(int), hipMemcpyHostToDevice);
    
    auto sort_start = std::chrono::steady_clock::now();
    thrust::sort_by_key(thrust::device, d_result, d_result+count_db_fingerprint, d_ids);
    auto sort_end = std::chrono::steady_clock::now();

    hipMemcpy(&result[0], d_result, count_db_fingerprint*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&ids[0], d_ids, count_db_fingerprint*sizeof(int), hipMemcpyDeviceToHost);
    
    /*for (int i=count_db_fingerprint-1 ; i>=0 ; i--) {
        std::cout << "ID " << ids[i] << "-"<< ids[i]/5 <<"\t: " << result[i];
        std::cout << std::endl;
    }*/
    auto timer_end = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff = timer_end - timer_start;
    std::chrono::duration<double> sort_time = sort_end - sort_start;
    std::cerr << "Time to get indexing result for " << count_db << " fingerprints in DB : " << diff.count()  << std::endl;
    std::cerr << "Time for sorting " << sort_time.count() << std::endl;

    hipFree(d_fp);
    hipFree(d_db);
    hipFree(d_result);
    hipFree(d_mapping);
    hipFree(d_s1_result);
    hipFree(d_s2_result);
    hipFree(d_s3_result);
    hipFree(d_s4_result);
    hipFree(d_ids);
    hipFree(d_s_sum);
    hipFree(d_cos_sum);
    hipFree(d_sin_sum);

    return 0;
}

// nvcc -o parallel_indexing_split_s1 parallel_indexing_split_s1.cu fingerprint_structure.cpp -std=c++11 -lineinfo
