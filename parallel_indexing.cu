#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <chrono>
#include "fingerprint_structure.hpp"
using namespace std;

const int BLOCKSIZE = 36;

// Constant weights
const float w1 = 0.16f;
const float w2 = 0.37f;
const float w3 = 0.16f;
const float w4 = 0.31f;

__host__ __device__ unsigned char dperiod_to_byte(float period) {
    float fresult = period/period_unit;
    unsigned char result = (char)fresult;
    return result;
}

__host__ __device__ float dbyte_to_period(unsigned char c) {
    float result = period_unit*(int)c;
    return result;
}

__host__ __device__ unsigned char dfrequency_to_byte(float frequency) {
    if (frequency == 0) {
        return dperiod_to_byte(frequency);
    } else {
        return dperiod_to_byte(1.0f/frequency);
    }
}

__host__ __device__ float dbyte_to_frequency(unsigned char c) {
    float result = dbyte_to_period(c);
    if (result == 0) return result;
    else return 1/result;
}

__device__ float dbyte_to_coherence(unsigned char c) {
    float result = (float)c/coherence_unit;
    return result;
}

__device__ float dbyte_to_orientation(unsigned char c) {
    float result = orientation_unit*(int)c;
    return result;
}

__global__ void calculate_s1(fingerprint* db, fingerprint* fp, float* result, int* mapping) {
    __shared__ float ss, scos, ssin;
    int j = blockIdx.x;
    int i = threadIdx.x;
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
        ss = 0;
        scos = 0;
        ssin = 0;
    }
    __syncthreads();
    // (db+j)->local_frequency[i] = dfrequency_t_byte(dbyte_to_frequency((db+j)->local_frequency[i])+0.1);
    float s = dbyte_to_coherence(fp->local_coherence[i])*dbyte_to_coherence((db+j)->local_coherence[i]);
    float d = M_PI/180.0f * 2 * (dbyte_to_orientation(fp->local_orientation[i])-dbyte_to_orientation((db+j)->local_orientation[i]));
    float tcos = s*cos(d);
    float tsin = s*sin(d);

    atomicAdd(&ss, s);
    atomicAdd(&scos, tcos);
    atomicAdd(&ssin, tsin);
    __syncthreads();
    if (i == 0) {
        result[j] = sqrt(pow(scos,2)+pow(ssin,2))/ss;
    }
    __syncthreads();
    // First core of a fingerprint check maximum from all core
    if (i == 0 && (db+j)->id%5 == 1) {
        int max_idx = j;
        for (int i=1 ; i<5 ; i++) {
            if ((db+j+1)->id%5 == 1) break;
            else {
                if (result[j+i] > result[max_idx]) {
                    max_idx = j+i;
                }
            }
        }
        mapping[((db+j)->id-1)/5] = max_idx;
    }
}

__global__ void calculate_s2(fingerprint* db, fingerprint* fp, float* result) {
    __shared__ float s_addition, s_absdiff;
    int j = blockIdx.x;
    int i = threadIdx.x;
    // int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float t_addition = dbyte_to_frequency(fp->local_frequency[i]) + dbyte_to_frequency((db+j)->local_frequency[i]);
    float t_absdiff = abs(dbyte_to_frequency(fp->local_frequency[i]) - dbyte_to_frequency((db+j)->local_frequency[i]));
    atomicAdd(&s_addition, t_addition);
    atomicAdd(&s_absdiff, t_absdiff);
    if (i == 0) {
        result[j] = 1 - (s_absdiff/s_addition);
    }
}

__global__ void calculate_s2_with_mapping(fingerprint* db, fingerprint* fp, float* result, int* mapping) {
    __shared__ float s_addition, s_absdiff;
    int j = mapping[blockIdx.x];
    int i = threadIdx.x;
    // int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float t_addition = dbyte_to_frequency(fp->local_frequency[i]) + dbyte_to_frequency((db+j)->local_frequency[i]);
    float t_absdiff = abs(dbyte_to_frequency(fp->local_frequency[i]) - dbyte_to_frequency((db+j)->local_frequency[i]));
    atomicAdd(&s_addition, t_addition);
    atomicAdd(&s_absdiff, t_absdiff);
    if (i == 0) {
        result[blockIdx.x] = 1 - (s_absdiff/s_addition);
    }
}

__global__ void calculate_s3(fingerprint* db, fingerprint* fp, float* result) {
    int j = blockIdx.x;
    result[j] = 1 - (abs(dbyte_to_frequency(fp->avg_frequency)-dbyte_to_frequency((db+j)->avg_frequency))/max(dbyte_to_frequency(fp->avg_frequency), dbyte_to_frequency((db+j)->avg_frequency)));
}

__global__ void calculate_s3_with_mapping(fingerprint* db, fingerprint* fp, float* result, int* mapping) {
    int j = mapping[blockIdx.x];
    result[blockIdx.x] = 1 - (abs(dbyte_to_frequency(fp->avg_frequency)-dbyte_to_frequency((db+j)->avg_frequency))/max(dbyte_to_frequency(fp->avg_frequency), dbyte_to_frequency((db+j)->avg_frequency)));
}

__global__ void calculate_s4(fingerprint* db, fingerprint* fp, float* result) {
    int j = blockIdx.x;
    result[j] = 1-(abs(dbyte_to_orientation(fp->avg_orientation)-dbyte_to_orientation((db+j)->avg_orientation))/M_PI);
}

__global__ void calculate_s4_with_mapping(fingerprint* db, fingerprint* fp, float* result, int* mapping) {
    int j = mapping[blockIdx.x];
    result[blockIdx.x] = 1-(abs(dbyte_to_orientation(fp->avg_orientation)-dbyte_to_orientation((db+j)->avg_orientation))/M_PI);
}

__global__ void calculate_s(float* s1, float* s2, float*s3, float* s4, float* result) {
    int i = threadIdx.x;
    result[i] = w1*s1[i] + w2*s2[i] + w3*s3[i] + w4*s4[i];
}

__global__ void calculate_s_with_mapping(float* s1, float* s2, float*s3, float* s4, float* result, int* mapping) {
    int i = threadIdx.x;
    result[i] = w1*s1[mapping[i]] + w2*s2[i] + w3*s3[i] + w4*s4[i];
}

__global__ void get_top_fingerprints(float* s, float* result, int* mapping) {
    int i = threadIdx.x;
    result[i] = s[mapping[i]];
}

int main(int argc, char** argv) {
    if (argc < 3) {
        cerr << "Usage : ./parallel_indexing fingerprint-to-be-searched fingerprint-db\n";
        return 0;
    }

    string fp_filename = argv[1];
    string db_filename = argv[2];
    cerr << "FP " << fp_filename << " DB " << db_filename << endl;

    // Read the fingerprint to be searched
    vector<struct fingerprint> fp;
    int count_fp = read_from_file(fp, fp_filename);

    vector<float> local_orie, local_cohe, local_freq;
    get_fingerprint_local_values(fp[0], local_orie, local_cohe, local_freq);
    float avg_o = get_fingerprint_average_orientation(fp[0]);
    float avg_f = get_fingerprint_average_frequency(fp[0]);

    // Read the database
    vector<struct fingerprint> db;
    int count_db = read_from_file(db, db_filename);
    cerr << "Fingerprint core database count : " << count_db << endl;

    cerr << "Last fingerprint ID : " << db[count_db-1].id << endl;
    int count_db_fingerprint = (db[count_db-1].id-1)/5+1;
    cerr << "Fingerprint database count : " << count_db_fingerprint << endl;

    auto timer_start = chrono::steady_clock::now();
    // Test S1
    fingerprint *d_fp, *d_db;
    float s1_result[count_db], s2_result[count_db], s3_result[count_db], s4_result[count_db];
    float result[count_db];
    float *d_result;

    hipMalloc((void **)&d_fp, sizeof(fingerprint));
    hipMalloc((void **)&d_db, count_db*sizeof(fingerprint));
    hipMalloc((void **)&d_result, count_db*sizeof(float));

    //Mapping for block idx to fingerprint core idx
    int *d_mapping;
    hipMalloc((void **)&d_mapping, count_db_fingerprint*sizeof(int));

    hipMemcpy(d_db, &db[0], count_db*sizeof(fingerprint), hipMemcpyHostToDevice);
    hipMemcpy(d_fp, &fp[0], sizeof(fingerprint), hipMemcpyHostToDevice);
    calculate_s1<<<count_db,BLOCKSIZE>>>(d_db, d_fp, d_result, d_mapping);
    hipMemcpy(&s1_result[0], d_result, count_db*sizeof(float), hipMemcpyDeviceToHost);

    int mapping[count_db_fingerprint];
    memset(mapping, 0, sizeof(mapping));
    hipMemcpy(&mapping[0], d_mapping, count_db_fingerprint*sizeof(int), hipMemcpyDeviceToHost);

    // for (int i=0 ; i<count_db ; i++) {
    //     cout << i << " : ID " << db[i].id << endl;
    //     cout << "result = " << s1_result[i] << endl;
    // }

    // Check mapping
    cout << "MAPPING\n";
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << i << " " << mapping[i] << endl;
    }

    // Test S2
    // Only calculate for 1 core per fingerprint
    // calculate_s2<<<count_db,BLOCKSIZE>>>(d_db, d_fp, d_result);
    calculate_s2_with_mapping<<<count_db_fingerprint,BLOCKSIZE>>>(d_db, d_fp, d_result, d_mapping);
    hipMemcpy(&s2_result[0], d_result, count_db*sizeof(float), hipMemcpyDeviceToHost);
    // cout << "\n\nS2\n";
    // for (int i=0 ; i<count_db ; i++) {
    //     cout << i << " : ID " << db[i].id << endl;
    //     cout << "result = " << s2_result[i] << endl;
    // }

    // Test S3
    // calculate_s3<<<count_db,1>>>(d_db, d_fp, d_result);
    calculate_s3_with_mapping<<<count_db_fingerprint,1>>>(d_db, d_fp, d_result,d_mapping);
    hipMemcpy(&s3_result[0], d_result, count_db*sizeof(float), hipMemcpyDeviceToHost);
    // cout << "\n\nS3\n";
    // for (int i=0 ; i<count_db ; i++) {
    //     cout << i << " : ID " << db[i].id << endl;
    //     cout << "result = " << s3_result[i] << endl;
    // }

    // Test S4
    // calculate_s4<<<count_db,1>>>(d_db, d_fp, d_result);
    calculate_s4_with_mapping<<<count_db_fingerprint,1>>>(d_db, d_fp, d_result, d_mapping);
    hipMemcpy(&s4_result[0], d_result, count_db*sizeof(float), hipMemcpyDeviceToHost);
    // cout << "\n\nS4\n";
    // for (int i=0 ; i<count_db ; i++) {
    //     cout << i << " : ID " << db[i].id << endl;
    //     cout << "result = " << s4_result[i] << endl;
    // }

    // Test S
    // Copy S1-S4 to device
    float *d_s1_result, *d_s2_result, *d_s3_result, *d_s4_result;
    hipMalloc((void **)&d_s1_result, count_db*sizeof(float));
    hipMalloc((void **)&d_s2_result, count_db*sizeof(float));
    hipMalloc((void **)&d_s3_result, count_db*sizeof(float));
    hipMalloc((void **)&d_s4_result, count_db*sizeof(float));
    hipMemcpy(d_s1_result, &s1_result[0], count_db*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s2_result, &s2_result[0], count_db*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s3_result, &s3_result[0], count_db*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s4_result, &s4_result[0], count_db*sizeof(float), hipMemcpyHostToDevice);
    // calculate_s<<<1,count_db>>>(d_s1_result, d_s2_result, d_s3_result, d_s4_result, d_result);
    calculate_s_with_mapping<<<1,count_db_fingerprint>>>(d_s1_result, d_s2_result, d_s3_result, d_s4_result, d_result, d_mapping);
    hipMemcpy(&result[0], d_result, count_db*sizeof(float), hipMemcpyDeviceToHost);
    cout << "\n\nS\n";
    // for (int i=0 ; i<count_db ; i++) {
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << i << " : ID " << db[i].id << endl;
        cout << "result = " << result[i] << endl;
    }

    float *d_final_result;
    /* This is for when not used with mapping */
    // get_top_fingerprints<<<1,count_db_fingerprint>>>(d_result, d_final_result, d_mapping);
    // hipMemcpy(&result[0], d_final_result, count_db_fingerprint*sizeof(float), hipMemcpyDeviceToHost);
    cout << "\n\nFinal Result\n";
    vector< pair<float, int> > best_matches;
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << "result = " << result[i] << endl;
        best_matches.push_back(make_pair(result[i], db[mapping[i]].id));
    }
    sort(best_matches.rbegin(), best_matches.rend());
    cout << "\nBest match\n";
    for (int i=0 ; i<best_matches.size() ; i++) {
        cout << "ID " << best_matches[i].second << "-"<< best_matches[i].second/5 <<"\t: " << best_matches[i].first << endl;
    }
    auto timer_end = chrono::steady_clock::now();
    chrono::duration<double> diff = timer_end - timer_start;
    cout << "Time to get indexing result for " << count_db << " fingerprints in DB : " << diff.count()  << endl;

    // DEBUG
    cout << "\nS1\n";
    for (int i=0 ; i<count_db ; i++) {
        cout << s1_result[i] << endl;
    }

    cout << "\nS1\n";
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << s1_result[mapping[i]] << endl;
    }

    cout << "\nS2\n";
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << s2_result[i] << endl;
    }

    cout << "\nS3\n";
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << s3_result[i] << endl;
    }

    cout << "\nS4\n";
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << s4_result[i] << endl;
    }

    cout << "\nS\n";
    for (int i=0 ; i<count_db_fingerprint ; i++) {
        cout << result[i] << endl;
    }

    hipFree(d_fp);
    hipFree(d_db);
    hipFree(d_result);
    hipFree(d_mapping);
    hipFree(d_s1_result);
    hipFree(d_s2_result);
    hipFree(d_s3_result);
    hipFree(d_s4_result);
    hipFree(d_final_result);

    return 0;
}

// nvcc -o parallel_indexing parallel_indexing.cu fingerprint_structure.cpp -std=c++11
